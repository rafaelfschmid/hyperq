/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "kernels.h"

void cudasafe(int error, char* message, char* file, int line) {
	if (error != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s : %i. In %s line %d\n", message, error,
				file, line);
		exit(-1);
	}
}

int getDeviceInformation() {
	hipDeviceProp_t deviceProp;

	cudasafe(hipGetDeviceProperties(&deviceProp, 0),
				"Get Device Properties", __FILE__, __LINE__);

	if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
		printf("No CUDA GPU has been detected\n");
	}
	else {
		printf("Device name:                %s\n", deviceProp.name);
		printf("Total Global Memory:        %d\n", deviceProp.regsPerMultiprocessor);
		printf("Total shared mem per block: %d\n", deviceProp.sharedMemPerMultiprocessor);
		printf("Total const mem size:       %d\n", deviceProp.maxThreadsPerMultiProcessor);
	}
}

int main(int argc, char **argv) {

	getDeviceInformation();



	return 0;
}
