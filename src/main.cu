/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <dlfcn.h>
#include "kernels.h"
#include "cudahook.h"

#include <functional>
#include <iostream>
#include <vector>
#include<tuple>
#include <future>

#include <thread>

typedef void* my_lib_t;

my_lib_t MyLoadLib(const char* szMyLib) {
	return dlopen(szMyLib, RTLD_LAZY);
}

void MyUnloadLib(my_lib_t hMyLib) {
	dlclose(hMyLib);
}

void* MyLoadProc(my_lib_t hMyLib, const char* szMyProc) {
	return dlsym(hMyLib, szMyProc);
}

typedef bool (*scheduleKernels_t)(int);
my_lib_t hMyLib = NULL;
scheduleKernels_t scheduleKernels = NULL;

bool callcudahook(int n) {
  if (!(hMyLib = MyLoadLib("/home/rafael/cuda-workspace/hyperq/src/libcudahook.so"))) { /*error*/ }
  if (!(scheduleKernels = (scheduleKernels_t)MyLoadProc(hMyLib, "scheduleKernels"))) { /*error*/ }

  bool ret = scheduleKernels(n);

  MyUnloadLib(hMyLib);

  return ret;
}

using namespace std;
using namespace std::placeholders;

void getDeviceInformation() {
	hipDeviceProp_t deviceProp;

	hipGetDeviceProperties(&deviceProp, 0);

	if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
		printf("No CUDA GPU has been detected\n");
	}
	else {
		printf("Device name:                %s\n", deviceProp.name);
		printf("Total Global Memory:        %d\n", deviceProp.regsPerMultiprocessor);
		printf("Total shared mem per block: %d\n", deviceProp.sharedMemPerMultiprocessor);
		printf("Total const mem size:       %d\n", deviceProp.maxThreadsPerMultiProcessor);
	}
}

//hipStream_t streams[NUM_STREAMS];

class Scheduler {
	std::vector<std::tuple<void (*)(uint, uint, uint, uint, hipStream_t), uint, uint, uint, uint>> functions;
	std::vector<int> map;
	int i=0;

public:
	hipStream_t *streams;
	int num_streams;

	Scheduler(int num_streams){
		this->num_streams = num_streams;
		streams = new hipStream_t[num_streams];
		for (int i = 0; i < this->num_streams; i++) {
			hipStreamCreate(&streams[i]);
		}
	}

	template<typename Func>
	void kernelCall(Func func, uint num_threads, uint num_blocks, uint shared_size, uint computation) {
		auto funct = make_tuple(func, num_threads, num_blocks, shared_size, computation);
		functions.push_back(funct);
		map.push_back(-1);
	}

	void schedule(){
		int k = 0;
		int j = 0;
		for(auto funct : functions){
			//printf("k=%d ", k);
			map[j++] = k;
			k=(++k) % num_streams;
		}
	}

	void execute(){
		int k = 0;

		//while(true) {
		//std::vector<std::thread> vec;
			std::vector<std::future<void>> vec;
			for(auto f : functions){
				//printf("\nk=%d", k);
				//auto a = std::async(std::launch::async, get<0>(f),get<1>(f),get<2>(f),get<3>(f),get<4>(f),streams[map[k]]);
				vec.push_back(std::async(std::launch::async, get<0>(f),get<1>(f),get<2>(f),get<3>(f),get<4>(f),streams[map[k]]));
				//vec.push_back(std::thread(get<0>(f),get<1>(f),get<2>(f),get<3>(f),get<4>(f),streams[map[k]]));
				k++;
			}
			//functions.clear();

			printf("testando0.1\n");
			callcudahook(vec.size());
			/*while(true){
				if (callcudahook(vec.size()))
					break;
			}*/

			printf("testando0.2\n");
			for(k = 0; k < vec.size(); k++){
				//printf("\nk=%d", k);
				//std::async(std::launch::async, get<0>(f),get<1>(f),get<2>(f),get<3>(f),get<4>(f),streams[map[k]]);
				//vec[k].join();
				vec[k].get();
				//printf("testando0.11\n");
				k++;
			}

		//}
		//printf("testando0.2\n");

		//executeKernels();
		//hipLaunchByPtr(NULL);
	}
};




int main(int argc, char **argv) {

	getDeviceInformation();

	Scheduler s(4);
	//callcudahook();


	uint num_threads = 16;
	uint num_blocks = 2;
	uint shared_size = 16;
	uint computation = 2;

	for(int i = 0; i < 5; i++) {
		s.kernelCall(kernel1, num_threads*i, num_blocks*i*2, shared_size*i, computation*i*1000);
	}

	s.schedule();
	printf("testando0\n");
	s.execute();
	printf("testando1\n");
	//callcudahook();
	//printf("testando2\n");

	return 0;
}
