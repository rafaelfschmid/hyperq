/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void cudasafe(int error, char* message, char* file, int line) {
	if (error != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s : %i. In %s line %d\n", message, error,
				file, line);
		exit(-1);
	}
}

void getDeviceInformation() {
	int deviceCount;
	cudasafe(hipGetDeviceCount(&deviceCount), "GetDeviceCount", __FILE__,
			__LINE__);

	printf("Number of CUDA devices %d.\n", deviceCount);

	for (int dev = 0; dev < deviceCount; dev++) {
		hipDeviceProp_t deviceProp;

		cudasafe(hipGetDeviceProperties(&deviceProp, dev),
				"Get Device Properties", __FILE__, __LINE__);

		if (dev == 0) {
			if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
				printf("No CUDA GPU has been detected\n");
				return -1;
			} else if (deviceCount == 1) {
				printf("There is 1 device supporting CUDA\n");
			} else {
				printf("There are %d devices supporting CUDA\n", deviceCount);
			}
		}

		printf("For device #%d\n", dev);
		printf("Device name:                %s\n", deviceProp.name);
		printf("Major revision number:      %d\n", deviceProp.major);
		printf("Minor revision Number:      %d\n", deviceProp.minor);
		printf("Total Global Memory:        %d\n", deviceProp.totalGlobalMem);
		printf("Total shared mem per block: %d\n",
				deviceProp.sharedMemPerBlock);
		printf("Total const mem size:       %d\n", deviceProp.totalConstMem);
		printf("Warp size:                  %d\n", deviceProp.warpSize);
		printf("Maximum block dimensions:   %d x %d x %d\n",
				deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
				deviceProp.maxThreadsDim[2]);

		printf("Maximum grid dimensions:    %d x %d x %d\n",
				deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
				deviceProp.maxGridSize[2]);
		printf("Clock Rate:                 %d\n", deviceProp.clockRate);
		printf("Number of multiprocessors:   %d\n",
				deviceProp.multiProcessorCount);

	}
}

int main(int argc, char **argv) {

	getDeviceInformation();

	return 0;
}
